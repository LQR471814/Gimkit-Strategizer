#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <iostream>
#include <stdlib.h>
#include <math.h>
#include "types.h"
#include "CLI11.hpp"

__host__ __device__ void printPlayState(PlayState p) {
	printf(
		"$%f Stats %d %d %d %d\n",
		(Money)p.money,
		p.stats.moneyPerQuestion,
		p.stats.streakBonus,
		p.stats.multiplier,
		p.stats.insurance
	);
}

__host__ __device__ UpgradeStats incrementStat(UpgradeStats s, int id)
{
	switch (id)
	{
	case MONEY_PER_QUESTION:
		s.moneyPerQuestion++;
		break;
	case STREAK_BONUS:
		s.streakBonus++;
		break;
	case MULTIPLIER:
		s.multiplier++;
		break;
	case INSURANCE:
		s.insurance++;
		break;
	}

	return s;
}

__host__ __device__ int getStat(UpgradeStats s, int id)
{
	switch (id)
	{
	case MONEY_PER_QUESTION:
		return s.moneyPerQuestion;
	case STREAK_BONUS:
		return s.streakBonus;
	case MULTIPLIER:
		return s.multiplier;
	case INSURANCE:
		return s.insurance;
	}

	return -1;
}

__host__ __device__ struct UpgradeLevel getUpgrade(UpgradeIndex *data, int id, int level)
{
	switch (id)
	{
	case MONEY_PER_QUESTION:
		return (*data).moneyPerQuestion[level];
	case STREAK_BONUS:
		return (*data).streakBonus[level];
	case MULTIPLIER:
		return (*data).multiplier[level];
	case INSURANCE:
		return (*data).insurance[level];
	}

	return UpgradeLevel{};
}

__host__ __device__ struct GoalResult playGoal(UpgradeIndex *data, PlayState s, Money goal, int giveup)
{
	int problems = 0;
	float streak = 0;
	Money money = s.money;

	UpgradeLevel mq = (*data).moneyPerQuestion[s.stats.moneyPerQuestion];
	UpgradeLevel sb = (*data).streakBonus[s.stats.streakBonus];
	UpgradeLevel mu = (*data).multiplier[s.stats.multiplier];
	UpgradeLevel in = (*data).insurance[s.stats.insurance];

	while (money < goal && (problems <= giveup || giveup < 0))
	{
	#ifdef __CUDA_ARCH__
		float i = hiprand_uniform(s.randState);
	#else
		float i = static_cast<float>(rand()) / static_cast<float>(RAND_MAX);
	#endif
		if (i < s.setbackChance)
		{
			money -= (Money) (mq.value * mu.value) - (mq.value * mu.value) * in.value / 100;
		}
		else
		{
			money += (Money) mu.value * (mq.value + sb.value * streak);
			streak++;
		}

		problems++;
	};

	return GoalResult{problems, money};
}

__host__ __device__ struct GoalResult playUpgrade(UpgradeIndex *data, PlayState s, int target, int giveup)
{
	if (getStat(s.stats, target) > data->MAX_LEVEL)
	{
		return GoalResult{0, s.money};
	};

	int goal = getUpgrade(data, target, getStat(s.stats, target) + 1).cost;
	GoalResult result = playGoal(data, s, goal, giveup);
	result.newMoney -= goal;

	return result;
}

struct std::vector<Permutation> permuteRecursive(PermuteContext *c, PermuteState r, int depth)
{
	if (depth == (*c).max)
	{
		Permutation p = Permutation{0, r.sequence, r.play};
		return std::vector<Permutation>{p};
	};

	std::vector<Permutation> permutes;
	for (int u : (*c).upgrades)
	{
		PlayState lowerState = r.play;
		GoalResult res = playUpgrade((*c).data, r.play, u, -1);
		lowerState.money = res.newMoney;
		lowerState.stats = incrementStat(r.play.stats, u);

		std::vector<int> lowerSequence;
		lowerSequence = r.sequence;
		lowerSequence.push_back(u);

		std::vector<Permutation> results = permuteRecursive(
			c, PermuteState{
				lowerState,
				lowerSequence,
			},
			depth + 1);

		for (Permutation p : results)
		{
			p.problems += res.problems;
			permutes.push_back(p);
		}
	}

	return permutes;
}

__host__ __device__ void printPlayStack(PlayStackFrame *stack, int depth, int resultLength) {
	printf("Depth %d Branch %d\n", depth, stack[depth].branch);
	printf(" Params\n");
	printf(" -> Problems %d\n", stack[depth].params.problems);
	printf(" -> ");
	printPlayState(stack[depth].params.state);
	printf(" -> Current Min %d Target %d\n", stack[depth].currentMin, stack[depth].minTarget);
}

__forceinline__ __host__ __device__ int iterativeCall(PlayStackFrame *stack, PlayStackParameters params, int depth)
{
	depth++;
	stack[depth].branch = 0;
	stack[depth].params = params;
	return depth;
}

__forceinline__ __host__ __device__ int iterativeReturn(PlayStackFrame *stack, int depth, int value)
{
	depth--;
	if (value < stack[depth].currentMin || stack[depth].currentMin < 0) {
		stack[depth].currentMin = value;
		stack[depth].minTarget = stack[depth].branch;
	};

	stack[depth].branch++;
	return depth;
}

__host__ __device__ int playIterative(RecurseContext *c, PlayState play, PlayStackFrame *stack, int *result, int startOffset)
{
	int depth = 0;
	stack[depth].params.state = play;

	//? To prevent crashes when the initial moneyValue is already larger than the goal
	if (stack[depth].params.state.money >= (*c).moneyGoal) {
		return 0;
	};

	while (true) {
		if (stack[depth].params.state.money >= (*c).moneyGoal) {
			depth = iterativeReturn(stack, depth, stack[depth].params.problems);
			continue;
		};

		if (stack[depth].branch == (*c).upgradesSize) {
			result[startOffset + depth] = stack[depth].minTarget;
			stack[depth].currentMin += stack[depth].params.problems;
			if (depth == 0) {
				return stack[depth].currentMin;
			};

			depth = iterativeReturn(stack, depth, stack[depth].currentMin);
			continue;
		};

		if (depth == (*c).max) {
			GoalResult res = playGoal((*c).data, stack[depth].params.state, (*c).moneyGoal, stack[depth].params.upperMinimum);
			depth = iterativeReturn(stack, depth, stack[depth].params.problems + res.problems);
			continue;
		};

		GoalResult res = playUpgrade(
			(*c).data, stack[depth].params.state,
			(*c).upgrades[stack[depth].branch],
			stack[depth].params.upperMinimum
		);

		PlayState lowerState = {
			incrementStat(
				stack[depth].params.state.stats,
				(*c).upgrades[stack[depth].branch]
			),
			stack[depth].params.state.setbackChance,
			res.newMoney,
			stack[depth].params.state.randState
		};

		depth = iterativeCall(stack, {
			lowerState,
			res.problems,
			stack[depth].currentMin
		}, depth);
	};
}

void assignVecToPointer(std::vector<int> vec, int *result, int size) {
	for (int i = 0; i < size; i++) {
		result[i] = vec[i];
	};
}

void allocUpgradeLevels(UpgradeLevel **results, std::vector<UpgradeLevel> levels) {
	hipMallocManaged(results, sizeof(UpgradeLevel) * levels.size());
	for (int i = 0; i < levels.size(); i++) {
		(*results)[i] = levels[i];
	};
}

std::vector<Permutation> getRoots(UpgradeIndex *data, std::vector<int> upgrades, int syncDepth) {
	PermuteContext c = {data, upgrades, syncDepth};
	PermuteState r = {
		UpgradeStats{0, 0, 0, 0}, // play
		0, 0, NULL,      		  // play
		std::vector<int>{},		  // sequence
	};

	return permuteRecursive(&c, r, 0);
}

struct UpgradeIndex* initializeIndex() {
	UpgradeIndex *data;
	hipMallocManaged(&data, sizeof(UpgradeIndex));
	data->MAX_LEVEL = index.MAX_LEVEL;

	allocUpgradeLevels(&(*data).moneyPerQuestion, moneyPerQuestionLevels);
	allocUpgradeLevels(&(*data).streakBonus, streakBonusLevels);
	allocUpgradeLevels(&(*data).multiplier, multiplierLevels);
	allocUpgradeLevels(&(*data).insurance, insuranceLevels);

	return data;
}

void deallocateIndex(UpgradeIndex *index) {
	hipFree((*index).moneyPerQuestion);
	hipFree((*index).streakBonus);
	hipFree((*index).multiplier);
	hipFree((*index).insurance);
	hipFree(index);
}

int* initializeSequence(std::vector<int> init, int targetSize) {
	int *sequence;
	hipMallocManaged(&sequence, sizeof(int) * targetSize);

	for (int i = 0; i < targetSize; i++) {
		sequence[i] = -1;
		if (i < init.size()) {
			sequence[i] = init[i];
		};
	};

	return sequence;
}

int* initializeUpgrades(std::vector<int> init) {
	int *upgrades;
	hipMallocManaged(&upgrades, sizeof(int) * init.size());
	for (int i = 0; i < init.size(); i++) {
		upgrades[i] = init[i];
	};

	return upgrades;
}

PlayStackFrame* initializeStack(int lowerDepth, int upgradesSize) {
	PlayStackFrame *stack;
	hipMallocManaged(&stack, sizeof(PlayStackFrame) * (lowerDepth+1));
	for (int i = 0; i < lowerDepth+1; i++) {
		stack[i] = {};
	};

	return stack;
}

__global__ void computeStrategy(RecurseContext *c, TRecurseResult *results, int rootSize, int depth)
{
	int index = threadIdx.x + blockIdx.x * blockDim.x;
	if (index >= rootSize) {
		return;
	};

	hiprand_init(1234, index, 0, results[index].init.randState);
	int problems = playIterative(
		c, results[index].init,
		results[index].stack,
		results[index].sequence, depth
	);

	results[index].problems = problems;
}

int computeSync(std::vector<int> upgrades, Money moneyGoal, int syncDepth, int maxDepth, int *result) {
	struct UpgradeIndex *data = initializeIndex();

	std::vector<Permutation> roots = getRoots(data, upgrades, syncDepth);
	int *recurseUpgrades = initializeUpgrades(upgrades);
	int lowerDepth = maxDepth - syncDepth;

	RecurseContext rc = {
		data,
		lowerDepth,
		moneyGoal,
		recurseUpgrades,
		static_cast<int>(upgrades.size()),
	};

	printf("Memory Allocation Succeeded\n");
	printf("Roots: %d\n", static_cast<int>(roots.size()));

	int min = -1;
	for (Permutation p : roots)
	{
		int *recurseResult = initializeSequence(p.sequence, maxDepth);
		PlayStackFrame *stack = initializeStack(lowerDepth, upgrades.size());

		printf("Problems: %d |", p.problems);
		for (int i = 0; i < p.sequence.size(); i++) {
			printf(" %d", p.sequence[i]);
		};
		printf("\n");

		int problems = p.problems + playIterative(&rc, p.play, stack, recurseResult, syncDepth);
		hipFree(stack);

		printf("Problems: %d |", problems);
		for (int i = 0; i < maxDepth; i++) {
			printf(" %d", recurseResult[i]);
		};
		printf("\n");

		if (min < 0 || problems < min) {
			min = problems;
			for (int i = 0; i < maxDepth; i++) {
				result[i] = recurseResult[i];
			};
		};

		hipFree(recurseResult);
	};

	hipFree(recurseUpgrades);
	return min;
}

int computeThreaded(std::vector<int> upgrades, Money moneyGoal, int syncDepth, int maxDepth, int *output) {
	struct UpgradeIndex *data = initializeIndex();
	std::vector<Permutation> roots = getRoots(data, upgrades, syncDepth);
	int *recurseUpgrades = initializeUpgrades(upgrades);

	int lowerDepth = maxDepth - syncDepth;

	RecurseContext c = {
		data,
		lowerDepth,
		moneyGoal,
		recurseUpgrades,
		static_cast<int>(upgrades.size()),
	};

	RecurseContext *rc = NULL;
	hipMallocManaged(&rc, sizeof(RecurseContext));
	*rc = c;

	TRecurseResult *results;
	hipMallocManaged(&results, sizeof(TRecurseResult) * roots.size());

	for (int i = 0; i < roots.size(); i++) {
		int *sequence = initializeSequence(roots[i].sequence, maxDepth);
		PlayStackFrame *stack = initializeStack(lowerDepth, upgrades.size());

		hiprandState *gen = NULL;
		hipMallocManaged(&gen, sizeof(hiprandState));

		roots[i].play.randState = gen;
		results[i] = TRecurseResult{
			roots[i].play,
			stack,
			roots[i].problems,
			sequence
		};
	};

	printf("Memory Allocation Succeeded\n");

	int threadBlocks = ceil(float(roots.size()) / float(BLOCK_SIZE));

	printf("Blocksize %d\n", BLOCK_SIZE);
	printf("Roots %zd Blocks %d\n", roots.size(), threadBlocks);
	computeStrategy<<<threadBlocks, BLOCK_SIZE>>>(
		rc, results, roots.size(), syncDepth
	);

	hipError_t err = hipDeviceSynchronize();
	printf("Compute Status %s\n", hipGetErrorString(err));

	int min = -1;
	for (int i = 0; i < roots.size(); i++) {
		int problems = roots[i].problems + results[i].problems;

		if (min < 0 || problems < min) {
			min = problems;
			for (int x = 0; x < maxDepth; x++) {
				output[x] = results[i].sequence[x];
			};
		};

		hipFree(results[i].init.randState);
		hipFree(results[i].sequence);
		hipFree(results[i].stack);
	};

	hipFree(results);
	hipFree(recurseUpgrades);
	hipFree(rc);
	deallocateIndex(data);
	return min;
}

int main(int argc, char** argv)
{
	CLI::App app{"A program that simulates many, many gimkit games"};

	std::string goalStr = "1000000";
	app.add_option(
		"-g,--goal",
		goalStr,
		"Amount of money to reach before stopping"
	);

	std::string syncStr = "5";
	app.add_option(
		"-r,--roots",
		syncStr,
		"The depth to recurse synchronously to (threads spawned = <amount of upgrades>^depth) (overrides block count)"
	);

	std::string depthStr = "10";
	app.add_option(
		"-d,--depth",
		depthStr,
		"The amount of upgrades to be purchased"
	);

	CLI11_PARSE(app, argc, argv);

	int syncDepth = std::stoi(syncStr);
	int maxDepth = std::stoi(depthStr);
	Money moneyGoal = std::stoi(goalStr);

	std::vector<int> upgrades = {
		MONEY_PER_QUESTION,
		STREAK_BONUS,
		MULTIPLIER};

	int *result = new int[maxDepth];
	int min = computeSync(upgrades, moneyGoal, syncDepth, maxDepth, result);
	// int min = computeThreaded(upgrades, moneyGoal, syncDepth, maxDepth, result);

	printf("========== RESULTS ==========\n");
	printf("Minimum Problems: %d\n", min);
	printf("Sequence Required: ");
	for (int i = 0; i < maxDepth; i++) {
		printf("%d ", result[i]);
	};

	return 0;
}
