#include "hip/hip_runtime.h"
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

#include <iostream>
#include <stdlib.h>
#include <math.h>

#include "types.hpp"
#include "CLI11.hpp"

#include "misc.hpp"
#include "pinned_memory.hpp"

__host__ __device__ struct GoalResult playGoal(UpgradeLevel **data, PlayState s, Money goal, int giveup)
{
	if (s.money >= goal) {
		return GoalResult{0, s.money};
	}

	float mq = data[MONEY_PER_QUESTION][s.stats.moneyPerQuestion].value;
	float sb = data[STREAK_BONUS][s.stats.streakBonus].value;
	float mu = data[MULTIPLIER][s.stats.multiplier].value;
	// float in = data[INSURANCE][s.stats.insurance].value;

	float a = mu*sb;
	float b = -mu*(sb - 2*mq);
	float c = 2*(s.money-goal);

	float problems = ceilf(
		(-b + sqrtf(pow(b, 2) - 4*a*c)) / (2*a)
	);

	Money money = s.money + (
		mu*problems * (
			2*mq + sb*(problems - 1)
		)
	) / 2;

	return GoalResult{int(problems), money};
}

__forceinline__ __host__ __device__ struct GoalResult playUpgrade(UpgradeLevel **data, PlayState s, int target, int giveup)
{
	int goal = data[target][getStat(s.stats, target) + 1].cost;
	GoalResult result = playGoal(data, s, goal, giveup);
	result.newMoney -= goal;

	return result;
}

struct std::vector<Permutation> permuteRecursive(PermuteContext *c, PermuteState r, int depth)
{
	if (depth == (*c).max)
	{
		Permutation p = Permutation{0, r.sequence, r.play};
		return std::vector<Permutation>{p};
	}

	std::vector<Permutation> permutes;
	for (int u : (*c).upgrades)
	{
		PlayState lowerState = r.play;
		GoalResult res = playUpgrade((*c).data, r.play, u, -1);
		lowerState.money = res.newMoney;
		lowerState.stats = incrementStat(r.play.stats, u);

		std::vector<int> lowerSequence;
		lowerSequence = r.sequence;
		lowerSequence.push_back(u);

		std::vector<Permutation> results = permuteRecursive(
			c, PermuteState{
				lowerState,
				lowerSequence,
			},
			depth + 1);

		for (Permutation p : results)
		{
			p.problems += res.problems;
			permutes.push_back(p);
		}
	}

	return permutes;
}

__host__ __device__ void printPlayStack(PlayStackFrame *stack, int depth, int resultLength) {
	printf("Depth %d Branch %d\n", depth, stack[depth].branch);
	printf(" Params\n");
	printf(" -> Problems %d\n", stack[depth].params.problems);
	printf(" -> ");
	printPlayState(stack[depth].params.state);
	printf(" -> Current Min %d Target %d\n", stack[depth].currentMin, stack[depth].minTarget);
}

__forceinline__ __host__ __device__ int iterativeCall(PlayStackFrame *stack, PlayStackParameters params, int depth)
{
	depth++;
	stack[depth].branch = 0;
	stack[depth].params = params;
	return depth;
}

__forceinline__ __host__ __device__ int iterativeReturn(PlayStackFrame *stack, int depth, int value)
{
	depth--;
	if (value < stack[depth].currentMin || stack[depth].currentMin < 0) {
		stack[depth].currentMin = value;
		stack[depth].minTarget = stack[depth].branch;
	}

	stack[depth].branch++;
	return depth;
}

__host__ __device__ int playIterative(ComputeContext *c, PlayState play, PlayStackFrame *stack, int *result, int startOffset)
{
	int depth = 0;
	stack[depth].params.state = play;

	//? To prevent crashes when the initial moneyValue is already larger than the goal
	if (stack[depth].params.state.money >= (*c).moneyGoal) {
		return 0;
	}

	while (true) {
		if (
			(*c).currentMinimum &&
			stack[depth].params.problems >= *(*c).currentMinimum &&
			*(*c).currentMinimum > 0
		) {
			depth = iterativeReturn(
				stack, depth,
				stack[depth].params.problems + 9999
			);
			continue;
		}

		if (stack[depth].params.state.money >= (*c).moneyGoal) {
			depth = iterativeReturn(
				stack, depth,
				stack[depth].params.problems
			);
			continue;
		}

		if (stack[depth].branch == (*c).upgradesSize) {
			result[startOffset + depth] = stack[depth].minTarget;
			if (depth == 0) {
				return stack[depth].currentMin;
			}

			depth = iterativeReturn(
				stack, depth,
				stack[depth].currentMin
			);
			continue;
		}

		if (depth == (*c).max) {
			GoalResult res = playGoal(
				(*c).data,
				stack[depth].params.state,
				(*c).moneyGoal,
				stack[depth].params.upperMinimum
			);

			depth = iterativeReturn(
				stack, depth,
				stack[depth].params.problems + res.problems
			);
			continue;
		}

		if (getStat(
			stack[depth].params.state.stats,
			(*c).upgrades[stack[depth].branch]
		)+1 == MAX_LEVEL) {
			depth = iterativeCall(stack, stack[depth].params, depth);
			continue;
		}

		GoalResult res = playUpgrade(
			(*c).data, stack[depth].params.state,
			(*c).upgrades[stack[depth].branch],
			stack[depth].params.upperMinimum
		);

		PlayState lowerState = {
			incrementStat(
				stack[depth].params.state.stats,
				(*c).upgrades[stack[depth].branch]
			),
			stack[depth].params.state.setbackChance,
			res.newMoney,
			stack[depth].params.state.randState
		};

		depth = iterativeCall(stack, {
			lowerState,
			stack[depth].params.problems + res.problems,
			stack[depth].currentMin
		}, depth);
	}
}

void assignVecToPointer(std::vector<int> vec, int *result, int size) {
	for (int i = 0; i < size; i++) {
		result[i] = vec[i];
	}
}

UpgradeLevel* allocUpgradeLevels(std::vector<UpgradeLevel> levels) {
	UpgradeLevel *result;
	hipMallocManaged(&result, sizeof(UpgradeLevel) * levels.size());
	for (int i = 0; i < levels.size(); i++) {
		result[i] = levels[i];
	}
	return result;
}

std::vector<Permutation> getRoots(UpgradeLevel **data, std::vector<int> upgrades, int syncDepth) {
	PermuteContext c = {data, upgrades, syncDepth};
	PermuteState r = {
		UpgradeStats{0, 0, 0, 0}, // play
		0, 0, NULL,      		  // play
		std::vector<int>{},		  // sequence
	};

	return permuteRecursive(&c, r, 0);
}

struct UpgradeLevel** initializeIndex() {
	UpgradeLevel **data;
	hipMallocManaged(&data, sizeof(UpgradeLevel) * UPGRADE_COUNT);

	data[MONEY_PER_QUESTION] = allocUpgradeLevels(moneyPerQuestionLevels);
	data[STREAK_BONUS] = allocUpgradeLevels(streakBonusLevels);
	data[MULTIPLIER] = allocUpgradeLevels(multiplierLevels);
	data[INSURANCE] = allocUpgradeLevels(insuranceLevels);

	return data;
}

void deallocateIndex(UpgradeLevel **data) {
	for (int i = 0; i < UPGRADE_COUNT; i++)
		hipFree(data[i]);
	hipFree(data);
}

int* initializeSequence(std::vector<int> init, int targetSize) {
	int *sequence;
	hipMallocManaged(&sequence, sizeof(int) * targetSize);

	for (int i = 0; i < targetSize; i++) {
		sequence[i] = -1;
		if (i < init.size()) {
			sequence[i] = init[i];
		}
	}

	return sequence;
}

int* initializeUpgrades(std::vector<int> init) {
	int *upgrades;
	hipMallocManaged(&upgrades, sizeof(int) * init.size());
	for (int i = 0; i < init.size(); i++) {
		upgrades[i] = init[i];
	}

	return upgrades;
}

PlayStackFrame* initializeStack(int lowerDepth, int upgradesSize) {
	PlayStackFrame *stack;
	hipMallocManaged(&stack, sizeof(PlayStackFrame) * (lowerDepth+1));
	for (int i = 0; i < lowerDepth+1; i++) {
		stack[i] = {};
	}

	return stack;
}

__global__ void computeStrategy(int *progress, ComputeContext *c, TComputeStates *results, int rootSize, int depth)
{
	int index = threadIdx.x + blockIdx.x * blockDim.x;
	if (index >= rootSize) {
		return;
	}

	hiprand_init(1234, index, 0, results[index].init.randState);
	int problems = playIterative(
		c, results[index].init,
		results[index].stack,
		results[index].sequence, depth
	);

	results[index].problems += problems;

	if (progress)
		atomicAdd(progress, 1);
}

ComputeContext* initializeContext(std::vector<int> upgrades, Money moneygoal, ComputeOptions options) {
	struct UpgradeLevel **data = initializeIndex();
	int *recurseUpgrades = initializeUpgrades(upgrades);

	int lowerDepth = options.maxDepth - options.syncDepth;
	int *globalMin;
	hipMallocManaged(&globalMin, sizeof(int));
	*globalMin = -1;

	ComputeContext c = {
		data,
		lowerDepth,
		moneygoal,
		recurseUpgrades,
		static_cast<int>(upgrades.size()),
		globalMin,
	};

	ComputeContext *rc = NULL;
	hipMallocManaged(&rc, sizeof(ComputeContext));
	*rc = c;

	return rc;
}

void deallocateContext(ComputeContext *rc) {
	hipFree(rc->upgrades);
	hipFree(rc->currentMinimum);
	deallocateIndex(rc->data);
	hipFree(rc);
}

TComputeStates* initializeThreadStates(std::vector<Permutation> roots, int upgrades, ComputeOptions opts) {
	TComputeStates *results;
	hipMallocManaged(&results, sizeof(TComputeStates) * roots.size());

	for (int i = 0; i < roots.size(); i++) {
		int *sequence = initializeSequence(roots[i].sequence, opts.maxDepth);
		PlayStackFrame *stack = initializeStack(opts.maxDepth - opts.syncDepth, upgrades);

		hiprandState *gen = NULL;
		hipMallocManaged(&gen, sizeof(hiprandState));

		roots[i].play.randState = gen;
		results[i] = TComputeStates{
			roots[i].play,
			stack,
			roots[i].problems,
			sequence
		};
	}

	return results;
}

int compute(std::vector<int> upgrades, Money moneyGoal, int *output, ComputeOptions opts) {
	// --> Initialize Roots / Compute States
	ComputeContext *rc = initializeContext(upgrades, moneyGoal, opts);
	std::vector<Permutation> roots = getRoots((*rc).data, upgrades, opts.syncDepth);
	TComputeStates* states = initializeThreadStates(roots, upgrades.size(), opts);
	printf("Memory Allocation Succeeded\n");

	int threadBlocks = ceil(float(roots.size()) / float(BLOCK_SIZE));
	printf("Blocksize %d\n", BLOCK_SIZE);
	printf("Roots %zd Blocks %d\n", roots.size(), threadBlocks);

	// --> Initialize progress and gpu compute control
	int *running = 0;
	int *progress = NULL, *d_progress = NULL;
	hipEvent_t start, stop;

	printGPUInfo();
	progress = createHostPointer<int>(0);
	d_progress = createPinnedPointer<int>(progress);

	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start);

	// --> Run
	computeStrategy<<<threadBlocks, BLOCK_SIZE>>>(
		d_progress, rc, states, roots.size(), opts.syncDepth
	);

	// --> Report progress
	hipEventRecord(stop);
	int bufProgress = 0;
	int trueProgress = 0;
	do {
		hipEventQuery(stop);
		trueProgress = *progress;
		if (trueProgress - bufProgress >= roots.size() * opts.loggingFidelity) {
			printf("Progress %d / %zd\n", bufProgress, roots.size());
			bufProgress = trueProgress;
		}
	} while (trueProgress < roots.size());

	// --> Report run performance
	hipEventSynchronize(stop);
	printf("\nCompute Status (ignore if there is no visible error) %s\n", hipGetErrorString(hipGetLastError()));

	float *elapsed = new float;
	hipEventElapsedTime(elapsed, start, stop);
	printf("Completed in %fs\n", (*elapsed) / 1000);
	delete elapsed;

	// --> Sort results
	int min = -1;
	for (int i = 0; i < roots.size(); i++) {
		if (min < 0 || states[i].problems < min) {
			min = states[i].problems;
			for (int x = 0; x < opts.maxDepth; x++) {
				output[x] = states[i].sequence[x];
			}
		}

		hipFree(states[i].init.randState);
		hipFree(states[i].sequence);
		hipFree(states[i].stack);
	}

	// --> Cleanup
	hipFree(states);
	deallocateContext(rc);
	return min;
}

int main(int argc, char** argv)
{
	CLI::App app{"A program that simulates many, many gimkit games"};

	Money moneyGoal = 1000000;
	app.add_option<Money, double>(
		"-g,--goal",
		moneyGoal,
		"Amount of money to reach before stopping"
	);

	unsigned int syncDepth = 2;
	app.add_option<unsigned int>(
		"-r,--roots",
		syncDepth,
		"The depth to recurse synchronously to (threads spawned = <amount of upgrades>^depth) (overrides block count)"
	);

	unsigned int maxDepth = 5;
	app.add_option<unsigned int>(
		"-d,--depth",
		maxDepth,
		"The amount of upgrades to be purchased"
	);

	float loggingFidelity = 0.05;
	app.add_option<float>(
		"-l,--logging-fidelity",
		loggingFidelity,
		"The fidelity in which progress is reported (smaller makes progress update more frequently)"
	);

	CLI11_PARSE(app, argc, argv);

	std::vector<int> upgrades = {
		MONEY_PER_QUESTION,
		STREAK_BONUS,
		MULTIPLIER,
	};

	ComputeOptions opts = {
		syncDepth,
		maxDepth,
		loggingFidelity,
	};

	int min = 0;
	int *result = new int[maxDepth];
	min = compute(upgrades, moneyGoal, result, opts);

	printf("========== RESULTS ==========\n");
	printf("Minimum Problems: %d\n", min);
	printf("Sequence Required: ");
	for (int i = 0; i < maxDepth; i++) {
		printf("%d ", result[i]);
	}

	return 0;
}
