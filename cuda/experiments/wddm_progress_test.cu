
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

__global__ void oof(int *progress)
{
	volatile int x = 0;
	while (x < 10000) {
		x += 17/23 + 2;
	};

	atomicAdd((int*)progress, 1);
}

int * createHostProgress() {
	int *progress;
	hipHostMalloc(&progress, sizeof(int), hipHostMallocDefault);
	hipHostRegister(progress, sizeof(int), 0);
	*progress = 0;

	return progress;
}

int * createPinnedProgress(int *hostPtr) {
	int *pinnedPtr;
	hipHostGetDevicePointer(&pinnedPtr, hostPtr, 0);
	return pinnedPtr;
}

int main()
{
	int threads = 256;

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	int *progress = createHostProgress();
	int *pinnedPtr = createPinnedProgress(progress);

	hipEventRecord(start);
	oof<<<1, threads>>>(pinnedPtr);
	hipEventRecord(stop);

	int hostProgress = 0;

	do {
		hipEventQuery(stop);
		int n = *progress;
		if (n - hostProgress >= threads * 0.1) {
			hostProgress = n;
			printf("Progress %d / %d\n", hostProgress, threads);
		};
	} while (hostProgress < threads);

	hipEventSynchronize(stop);

	printf("Done!\n");
	return 0;
}