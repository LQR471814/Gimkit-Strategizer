#include "hip/hip_runtime.h"
#include <iostream>
#include <math.h>
#include <hip/hip_runtime.h>
#include <hiprand.h>

// Kernel function to add the elements of two arrays
__global__
void multiply(int n, float *r)
{
	hiprandGenerator_t *g;
	hipMallocManaged(&g, sizeof(hiprandGenerator_t));

	hiprandCreateGenerator(g, HIPRAND_RNG_PSEUDO_DEFAULT);
	hiprandGenerateUniform(*g, r, n);
}

int main(void)
{
	// int threads = 256;
	int N = 1<<20;
	float *r;

	// Allocate Unified Memory – accessible from CPU or GPU
	hipMallocManaged(&r, N*sizeof(float));

	// Run kernel on 1M elements on the GPU
	multiply<<<1, 1>>>(N, r);

	// Wait for GPU to finish before accessing on host
	hipDeviceSynchronize();

	// Check for errors (all values should be 3.0f)
	for (int i = 0; i < N; i++)
		std::cout << printf("%f", r[i]) << std::endl;

	// Free memory
	hipFree(r);

	return 0;
}
