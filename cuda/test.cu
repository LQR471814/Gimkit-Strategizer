#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <iostream>
#include <stdlib.h>
#include <math.h>
#include "types.h"

struct ScamResult {
	int someNumber;
	float *result;
	hiprandState *randGen;
};

__global__ void thread(int *data, ScamResult *result) {
	hiprand_init(1234, threadIdx.x, 0, result[threadIdx.x].randGen);
	*result[threadIdx.x].result = hiprand_uniform(result[threadIdx.x].randGen);
}

int main(int argc, char** argv) {
	printf("Arguments %d\n", argc);

	for (int i = 1; i < argc; i++) {
		printf("Argument Index %d Value %s\n", argc, argv[i]);
	};

	int threads = 32;

	int *data;
	hipMallocManaged(&data, sizeof(int));
	*data = 3;

	ScamResult *res;
	hipMallocManaged(&res, sizeof(ScamResult) * threads);
	for (int i = 0; i < threads; i++) {
		float *result;
		hipMallocManaged(&result, sizeof(int));

		hiprandState *gen;
		hipMallocManaged(&gen, sizeof(hiprandState));

		res[i] = ScamResult{
			0, result, gen
		};
	};

	thread<<<1, threads>>>(data, res);
	hipDeviceSynchronize();

	for (int i = 0; i < threads; i++) {
		printf("result %f\n", *res[i].result);
	};

	return 0;
}