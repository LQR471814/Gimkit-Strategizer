#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <iostream>
#include <stdlib.h>
#include <math.h>
#include "types.h"

struct ScamResult {
	int someNumber;
	float *result;
	hiprandState *randGen;
};

__forceinline__ __device__ int process(int *yo, int ack, int value) {
	ack++;
	yo[ack] = value;
	return ack;
}

__global__ void thread(int *data, ScamResult *result) {
	hiprand_init(1234, threadIdx.x, 0, result[threadIdx.x].randGen);
	*result[threadIdx.x].result = hiprand_uniform(result[threadIdx.x].randGen);

	int *yo = new int[10];
	yo[0] = 1234;
	printf("%d\n", yo[0]);

	int ok = process(yo, 0, 4321);
	printf("Ok %d ACKK %d\n", ok, yo[ok]);
}

int main(int argc, char** argv) {
	printf("Arguments %d\n", argc);

	for (int i = 1; i < argc; i++) {
		printf("Argument Index %d Value %s\n", argc, argv[i]);
	};

	int threads = 32;

	int *data;
	hipMallocManaged(&data, sizeof(int));
	*data = 3;

	ScamResult *res;
	hipMallocManaged(&res, sizeof(ScamResult) * threads);
	for (int i = 0; i < threads; i++) {
		float *result;
		hipMallocManaged(&result, sizeof(int));

		hiprandState *gen;
		hipMallocManaged(&gen, sizeof(hiprandState));

		res[i] = ScamResult{
			0, result, gen
		};
	};

	thread<<<1, threads>>>(data, res);
	hipDeviceSynchronize();

	for (int i = 0; i < threads; i++) {
		printf("result %f\n", *res[i].result);
	};

	return 0;
}